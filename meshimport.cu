
#include <hip/hip_runtime.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <map>
#include <sstream>
#include <string>
#include <vector>

// Data Structures
struct Node
{
  int index;
  double x, y, z;
};

struct Element
{
  int index;
  std::vector<int> nodeIndices;
};

struct FlatElement
{
  int index;
  int *nodeIndices;
  int numIndices;
};

struct FlatNodeSet
{
  int *nodeIndices;
  int numIndices;
};

struct NodeSet
{
  std::string name;
  std::vector<int> nodeIndices;
};

// Utility Functions
std::string trim(const std::string &str)
{
  size_t first = str.find_first_not_of(' ');
  if (std::string::npos == first)
  {
    return str;
  }
  size_t last = str.find_last_not_of(' ');
  return str.substr(first, (last - first + 1));
}

// Function to convert elements to a flat structure
void convertElementsToFlat(const std::map<int, Element> &elementsMap, FlatElement **flatElements, int *totalNumIndices)
{
  int numElements = elementsMap.size();
  hipMallocManaged(flatElements, numElements * sizeof(FlatElement));

  int indexCounter = 0;
  for (const auto &elem : elementsMap)
  {
    (*flatElements)[indexCounter].index = elem.second.index;
    (*flatElements)[indexCounter].numIndices = elem.second.nodeIndices.size();
    hipMallocManaged(&((*flatElements)[indexCounter].nodeIndices), elem.second.nodeIndices.size() * sizeof(int));

    for (size_t i = 0; i < elem.second.nodeIndices.size(); ++i)
    {
      (*flatElements)[indexCounter].nodeIndices[i] = elem.second.nodeIndices[i];
    }
    *totalNumIndices += elem.second.nodeIndices.size();
    ++indexCounter;
  }
}

// Similarly for nodeSets
void convertNodeSetsToFlat(const std::map<std::string, NodeSet> &nodeSetsMap, FlatNodeSet **flatNodeSets, int *totalNumIndices)
{
  int numNodeSets = nodeSetsMap.size();
  hipMallocManaged(flatNodeSets, numNodeSets * sizeof(FlatNodeSet));

  int indexCounter = 0;
  for (const auto &set : nodeSetsMap)
  {
    (*flatNodeSets)[indexCounter].numIndices = set.second.nodeIndices.size();
    hipMallocManaged(&((*flatNodeSets)[indexCounter].nodeIndices), set.second.nodeIndices.size() * sizeof(int));

    for (size_t i = 0; i < set.second.nodeIndices.size(); ++i)
    {
      (*flatNodeSets)[indexCounter].nodeIndices[i] = set.second.nodeIndices[i];
    }
    *totalNumIndices += set.second.nodeIndices.size();
    ++indexCounter;
  }
}

// Main Parsing Logic
int main(int argc, char *argv[])
{
  if (argc != 2)
  {
    std::cerr << "Usage: " << argv[0] << " <mesh_file_path>" << std::endl;
    return 1;
  }

  std::string meshFilePath = argv[1];
  std::ifstream meshFile(meshFilePath);
  if (!meshFile.is_open())
  {
    std::cerr << "Failed to open file: " << meshFilePath << std::endl;
    return 1;
  }

  std::string line;

  std::vector<Node> nodes;
  std::map<int, Element> elements;
  std::map<std::string, NodeSet> nodeSets;

  bool inNodesSection = false, inElementsSection = false,
       inNodeSetsSection = false;
  std::string currentSetName;

  while (getline(meshFile, line))
  {
    line = trim(line);
    if (line.empty() || line[0] == '*')
    {
      inNodesSection = line.find("*Node") != std::string::npos;
      inElementsSection = line.find("*Element") != std::string::npos;
      inNodeSetsSection = line.find("*Nset") != std::string::npos;

      if (inNodeSetsSection)
      {
        size_t namePos = line.find("Nset=");
        if (namePos != std::string::npos)
        {
          currentSetName = line.substr(namePos + 5);
          nodeSets[currentSetName] = NodeSet{currentSetName};
        }
      }

      continue;
    }

    if (inNodesSection)
    {
      std::istringstream iss(line);
      std::string indexStr;
      std::getline(iss, indexStr,
                   ',');                   // Read up to the first comma to get the node index.
      int nodeIndex = std::stoi(indexStr); // Convert index string to int.

      Node node;
      node.index = nodeIndex;

      std::string coordinateStr;
      std::getline(iss, coordinateStr,
                   ',');                 // Read up to the next comma for the x coordinate.
      node.x = std::stod(coordinateStr); // Convert to double.

      std::getline(iss, coordinateStr,
                   ',');                 // Read up to the next comma for the y coordinate.
      node.y = std::stod(coordinateStr); // Convert to double.

      std::getline(iss,
                   coordinateStr);       // Read the rest of the line for the z
                                         // coordinate (assuming no more commas).
      node.z = std::stod(coordinateStr); // Convert to double.

      nodes.push_back(node);
    }
    else if (inElementsSection)
    {
      std::istringstream iss(line);
      Element element;
      if (!(iss >> element.index))
      { // Read and check the element's index.
        std::cerr << "Failed to read element index from line: " << line
                  << std::endl;
        continue; // Skip to the next line if the element index can't be read.
      }

      // Read the rest of the line as a single string.
      std::string restOfLine;
      std::getline(iss, restOfLine);

      // Use another stringstream to parse the node indices from restOfLine.
      std::istringstream nodeStream(restOfLine);
      std::string
          nodeIndexStr; // Use a string to temporarily hold each node index.

      while (std::getline(nodeStream, nodeIndexStr,
                          ','))
      { // Read up to the next comma.
        if (!nodeIndexStr.empty())
        { // Check if the string is not empty.
          std::istringstream indexStream(
              nodeIndexStr); // Use another stringstream to convert string to
                             // int.
          int nodeIndex;
          if (indexStream >> nodeIndex)
          { // Convert the string to an int.
            element.nodeIndices.push_back(nodeIndex);
          }
        }
      }
      elements[element.index] = element;
    }
    else if (inNodeSetsSection && !currentSetName.empty())
    {
      std::istringstream iss(line);
      int nodeIndex;
      while (iss >> nodeIndex)
      {
        nodeSets[currentSetName].nodeIndices.push_back(nodeIndex);
      }
    }
  }

  meshFile.close();

  // Convert elements and nodeSets to flat structures
  FlatElement *flatElements = nullptr;
  FlatNodeSet *flatNodeSets = nullptr;
  int totalElementIndices = 0, totalNodeSetIndices = 0;

  convertElementsToFlat(elements, &flatElements, &totalElementIndices);
  convertNodeSetsToFlat(nodeSets, &flatNodeSets, &totalNodeSetIndices);

  std::ofstream outputFile(
      "debug_output.txt"); // Open a file for writing debug information

  if (!outputFile.is_open())
  {
    std::cerr << "Failed to open debug output file." << std::endl;
    return 1; // Exit if the file cannot be opened
  }

  // Output parsed nodes data to the file
  outputFile << "Parsed Nodes:" << std::endl;
  for (const auto &node : nodes)
  {
    outputFile << "Node " << node.index << ": (" << node.x << ", " << node.y
               << ", " << node.z << ")" << std::endl;
  }

  // Output parsed elements data to the file
  outputFile << "\nParsed Elements:" << std::endl;
  for (const auto &pair : elements)
  {
    outputFile << "Element " << pair.first << ":";
    for (int ni : pair.second.nodeIndices)
    {
      outputFile << " " << ni;
    }
    outputFile << std::endl;
  }

  // Output parsed node sets data to the file
  outputFile << "\nParsed Node Sets:" << std::endl;
  for (const auto &pair : nodeSets)
  {
    outputFile << "Node Set " << pair.first << ":";
    for (int ni : pair.second.nodeIndices)
    {
      outputFile << " " << ni;
    }
    outputFile << std::endl;
  }

  // Close the output file
  outputFile.close();

  return 0;
}
